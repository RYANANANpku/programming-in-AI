
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <memory>
#include <vector>
#include <cmath>
#include <hipblas.h>

//C(m,n) = A(m,k) * B(k,n)
void gemm_gpu(const float *A, const float *B, float *C, const int m, const int k, const int n) 
{
    int lda = k, ldb = k, ldc = m;
    const float alf = 1, bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
    // Create a handle for CUBLAS
    hipblasHandle_t handle; hipblasCreate(&handle);
    // Do the actual multiplication
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
    // Destroy the handle
    hipblasDestroy(handle);
}



int main() {
    int m = 3; 
    int n = 2; 
    int k = 4; 

    std::vector<float> A = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0, 11.0, 12.0};
    std::vector<float> B = {13.0, 14.0, 15.0, 16.0, 17.0, 18.0, 19.0, 20.0, 21.0, 22.0, 23.0, 24.0};
    std::vector<float> C(m * n, 0.0);

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, A.size() * sizeof(float));
    hipMalloc((void**)&d_B, B.size() * sizeof(float));
    hipMalloc((void**)&d_C, C.size() * sizeof(float));

    hipMemcpy(d_A, A.data(), A.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), B.size() * sizeof(float), hipMemcpyHostToDevice);

    gemm_gpu(d_A, d_B, d_C, m, k, n);

    hipMemcpy(C.data(), d_C, C.size() * sizeof(float), hipMemcpyDeviceToHost);

    
    std::cout << "output matrix: C:" << std::endl;
    for (int i = 0; i < n*m; ++i) {
        std::cout << C[i] << " ";
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
