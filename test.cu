
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <memory>
#include <vector>
#include <cmath>
#include <hipblas.h>
#include <hiprand/hiprand.h>

// Use 512 threads per block
const int kCudaThreadsNum = 512;
inline int CudaGetBlocks(const int N) {
   return (N + kCudaThreadsNum - 1) / kCudaThreadsNum;
}

#define CUDA_KERNEL_LOOP(i, n)                         \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;  \
       i < (n);                                        \
       i += blockDim.x * gridDim.x)

/* in this file, we are going to test the correctness of our functions */

/* matrix generator,we use curand to generate a matrix */
void matrix_init(float*A, int rows, int cols){
    // Create a pseudo-random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
    // Set the seed for the random number generator using the system clock
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());
    // Fill the array with random numbers on the device
    hiprandGenerateUniform(prng, A, rows * cols);
    hiprandDestroyGenerator(prng);
}

void matrix_print(float* data, int size)
{
    for(int i = 0; i < size; i++)
    {
        std::cout << data[i] << " ";
    }
    std::cout << std::endl;
    return;
}

/* first: fully connected layer */

/* before the impletation of fully connected layer,we have to make some assumptions
   we assume that the input X is in_features * batch_size
   we assume that the weight W is out_features * in_features
   we assume that the output Y is out_features * batch_size
*/
//C(m,n) = A(m,k) * B(k,n)
void gemm_gpu(hipblasOperation_t op1, hipblasOperation_t op2, const float *A, const float *B, float *C, const int m, const int k, const int n, float p1, float p2) 
{
    int lda,ldb,ldc = m;
    if(op1 == HIPBLAS_OP_N) lda = m;
    else lda = k;
    if(op2 == HIPBLAS_OP_N) ldb = k;
    else ldb = n;
    const float alf = p1, bet = p2;
    const float *alpha = &alf;
    const float *beta = &bet;
    // Create a handle for CUBLAS
    hipblasHandle_t handle; hipblasCreate(&handle);
    // Do the actual multiplication
    hipblasSgemm(handle, op1, op2, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
    // Destroy the handle
    hipblasDestroy(handle);
}

void forward_fc(float* input, float* output, float* weights, float* bias, int batch_size, int in_features, int out_features) 
{
    // matrix product with gemm
    gemm_gpu(HIPBLAS_OP_N, HIPBLAS_OP_N, weights, input, output, out_features, in_features, batch_size, 1.0 , 0.0);
    
    // add bias
    std::vector<float> _ones(batch_size, 1.0);
    float *d_ones;
    hipMalloc((void**)&d_ones, _ones.size() * sizeof(float));
    hipMemcpy(d_ones, _ones.data(), _ones.size() * sizeof(float), hipMemcpyHostToDevice);
    gemm_gpu(HIPBLAS_OP_N, HIPBLAS_OP_N, bias, d_ones, output, out_features, 1, batch_size, 1.0 , 1.0);
    
    return;
}

/* grad_output -- y (out_features*batch_size)  grad_input -- x (in_features * batch_size) 
   grad_weights -- out_features * in_features*/
void backward_fc(float* input, float* output, float* weights, float* bias, int batch_size, int in_features, int out_features,
                 float* grad_output, float* grad_input, float* grad_weights, float* grad_bias)
{
    // compute grad_input
    gemm_gpu(HIPBLAS_OP_T, HIPBLAS_OP_N, weights, grad_output, grad_input, in_features, out_features, batch_size, 1.0 , 0.0);
    // compute grad_weight
    gemm_gpu(HIPBLAS_OP_N, HIPBLAS_OP_T, grad_output, input, grad_weights, out_features, batch_size, in_features, 1.0 , 0.0);
    // compute grad_bias?
    hipMemcpy(grad_bias, grad_output, out_features*batch_size*sizeof(float), hipMemcpyDeviceToDevice);
    return;
}

void im2col(const float* data_im, float* data_col, int H, int W, int C, int N)
{
    int k_size = 3;
    //int stride = 1;
    //int padding = 1;
    /* we consider the last situation: input is N*C*H*W */
    //int space = N*C*H*W*k_size*k_size*sizeof(float);
    //data_col = (float*)malloc(sizeof(space));
    int length = H*W*N;
    int width = C*k_size*k_size;
    for(int col_i = 0; col_i < length; col_i ++)
    {
        for(int col_j = 0; col_j < width; col_j ++)
        {
            int batch_num = col_i/(H*W);
            int w_i = (col_i%(H*W))/W;
            int w_j = (col_i%(H*W))%W;
            int c_num = col_j/(k_size*k_size);
            int d_i = (col_j%(k_size*k_size))/3 - 1;
            int d_j = (col_j%(k_size*k_size))%3 - 1;
            int im_i = w_i + d_i;
            int im_j = w_j + d_j;
            if(im_i < 0 || im_j < 0 || im_i >= H || im_j >= W) data_col[col_i*width + col_j] = 0;
            else data_col[col_i*width + col_j] = data_im[batch_num*H*W*C + c_num*H*W + im_i*W + im_j];
        }
    }
    return;
}

int main()
{
    /* batchsize:3 , C_in:2 , C_out:4 . So we can construct our input X (2,3) and weight matrix W (4,2) */
    std::vector<float> X_cpu = {1.0,-2.0, 4.0,-3.0, 5.0,7.0};
    std::vector<float> W_cpu = {-2.0,1.0,3.0,4.0,-2.0,3.0,4.0,6.0};
    std::vector<float> bias = {1.0,-1.0,0,1.0};
    float *X, *W, *B;
    hipMalloc(&X,6*sizeof(float));
    hipMalloc(&W,8*sizeof(float));
    hipMalloc(&B,4*sizeof(float));

    hipMemcpy(X,X_cpu.data(),6*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(W,W_cpu.data(),8*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(B,bias.data(),4*sizeof(float),hipMemcpyHostToDevice);

    float* output = nullptr;
    hipMalloc(&output, 12*sizeof(float));

    forward_fc(X,output,W,B,3,2,4);

    float* Y_cpu = (float*)malloc(12*sizeof(float));
    hipMemcpy(Y_cpu,output,12*sizeof(float),hipMemcpyDeviceToHost);
    //matrix_print(Y_cpu,12);

    /* we let the output to be the grad_output, so that we can test our backward function */
    float *grad_input, *grad_weights, *grad_bias;
    hipMalloc(&grad_input, 6*sizeof(float));
    hipMalloc(&grad_weights, 8*sizeof(float));
    hipMalloc(&grad_bias,12*sizeof(float));

    backward_fc(X,output,W,B,3,2,4,output,grad_input,grad_weights,grad_bias);

    float *grad_input_cpu, *grad_weights_cpu;
    grad_input_cpu = (float*)malloc(6*sizeof(float));
    grad_weights_cpu = (float*)malloc(8*sizeof(float));
    hipMemcpy(grad_input_cpu,grad_input,6*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(grad_weights_cpu,grad_weights,8*sizeof(float),hipMemcpyDeviceToHost);

    //matrix_print(grad_input_cpu,6);
    //matrix_print(grad_weights_cpu,8);

    /* next: test the convolution layer */
    /* test the im2col function first, we assume that the batch_size is 2,the channel is 3,the (H,W) is (5,4) */
    float *data = (float*)malloc(4*5*3*2*sizeof(float));
    for(int i = 0; i < 120; i++)
    {
        data[i] = i;
    }
    float* data_col = (float*)malloc(5*4*3*2*3*3);
    im2col(data, data_col, 5, 4, 3, 2);
    for(int i = 0; i < 40; i ++)
    {
        for(int j = 0; j < 27; j++)
        {
            std::cout << data_col[i*27 + j] << " ";
        }
        std::cout << std::endl;
    }


    return 0;
}